#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <string>
#include "globals.hpp"

// nvcc -arch=sm_75 -o output_file source_file.cu

__device__ float genConProb(int i, int j) {

  int pres_pop = which_pop[j];
  int post_pop = which_pop[i];

  float theta_i = (2.0 * M_PI * i) / (float) Na[post_pop];
  float theta_j = (2.0 * M_PI * j) / (float) Na[pres_pop];

  float proba = Ka[pres_pop] / Na[pres_pop];

  if (PROBA == "cos")
    proba *= (1.0 + KAPPA[pres_pop + N_POP * post_pop] * cos(theta_i - theta_j));
  else if (PROBA == "spec")
    proba *= (1.0 + KAPPA[pres_pop + N_POP * post_pop] / sqrt(Ka[pres_pop]) * cos(theta_i - theta_j));

  return proba;
}

__global__ void genSparseMatCSC(unsigned long* colptr, int* indices) {

  int j = blockIdx.x * blockDim.x + threadIdx.x;

  // Each CUDA thread generates connections for multiple pre-synaptic neurons
  for (int k = threadIdx.y; k < N; k += blockDim.y) {
    int i = k;
    float P = genConProb(i, j, which_pop, Na, Ka, N_POP, KAPPA, PROBA);
    int nnz = 0;
    if (P > 0) {
      nnz = atomicAdd(&colptr[j+1], 1);
      indices[colptr[j] + nnz] = i;
    }

    // Generate connections for the rest of the post-synaptic neurons
    for (i = k+blockDim.y; i < N; i += blockDim.y) {
      P = genConProb(i, j, which_pop, Na, Ka, N_POP, KAPPA, PROBA);
      if (P > 0) {
        nnz = atomicAdd(&colptr[j+1], 1);
        indices[colptr[j] + nnz] = i;
      }
    }
  }

  // Write NNZ to colptr[0] once all the threads have completed
  __syncthreads();

  if (threadIdx.y == 0 && threadIdx.x == 0) {
    unsigned long total_nnz = colptr[blockDim.x * blockDim.y];
    colptr[0] = 0;
    for (int i = 1; i <= N; i++) {
      unsigned long tmp = colptr[i];
      colptr[i] = colptr[i-1] + total_nnz;
      total_nnz = tmp;
    }
  }
}

int main() {
  
  std::string configname = argv[1] ;
  loadConfig(configname);
  
  dim3 blockSize(32, 8);  // 256 threads
  dim3 gridSize((N + blockSize.x - 1) / blockSize.x, 1);  // 1D grid
  
  genSparseMatCSC<<<gridSize, blockSize>>>(colptr, indices);
  
  hipDeviceSynchronize();  // Wait for kernel to finish
  
}
