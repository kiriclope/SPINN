#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "globals.hpp"
#include "utils.hpp"

__global__ void initCurand(hiprandState* states, unsigned long seed)
{
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ float genConProb(int i, int j, int Na, int Nb, float Kb, float kappa) {
  
  float theta_i = (2.0 * M_PI * i) / (float) Na;
  float theta_j = (2.0 * M_PI * j) / (float) Nb;
  
  float proba = Kb / (float) Nb;
  proba *= (1.0 + kappa * cos(theta_i - theta_j));
  
  return proba;
}

__global__ void genSparseMatKernel(unsigned long long* nnz, size_t* colptr, int* indices, hiprandState* states, int Na, int Nb, float Kb, float kappa) {
  
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (j < Nb) {
    float rand_num;
    float proba;
    unsigned long long local_nnz = 0;
    
    colptr[j] = *nnz;
    for (int i = 0; i < Na; i++) { // postsynaptic
      rand_num = hiprand_uniform(&states[j]);
      
      proba = genConProb(i, j, Na, Nb, Kb, kappa);
      
      if (rand_num < proba) {
        indices[*nnz + local_nnz] = i;
        local_nnz++;
      }
    }
    colptr[j+1] = local_nnz + *nnz ;
    atomicAdd(nnz, local_nnz);
  }
}

void genSparseMatCSC(size_t*& colptr, int*& indices, int Na, int Nb, float Kb, float kappa, size_t offset) {
  
  std::cout << "Generating Sparse Matrix" ;
  
  // Allocate memory on the device
  unsigned long long* nnz=0, nnzz=0;
  hipMallocManaged(&nnz, sizeof(*nnz));
  *nnz = 0; // important: initialize since atomicAdd is an incrementing operation
  
  size_t *d_colptr;
  int *d_indices;
  hiprandState *d_states;  
  
  for(int i_pop=0; i_pop<2; ++i_pop) {
    hipMalloc(&d_colptr, (Nb+1) * sizeof(size_t));
    hipMalloc(&d_indices, Na * Nb * sizeof(int));
    
    // Allocate space for hiprand states
    hipMalloc(&d_states, Nb * sizeof(hiprandState));
  
    // Initialize hiprand
    initCurand<<<Nb, 1>>>(d_states, i_pop*4);

    nnzz = *nnz;
    // Invoke the kernel    
    genSparseMatKernel<<<Nb, 1>>>(nnz, d_colptr, d_indices, d_states, Na, Nb, Kb, kappa);
    hipDeviceSynchronize(); // make sure all device operations finish before accessing data on the host
    
    std::cout << " nnz " << *nnz << std::endl; 
    // hipFree(nnz);
    
    // Copy result back to host
    hipMemcpy(&colptr[offset], d_colptr, (Nb+1) * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&indices[nnzz], d_indices, Na * Nb * sizeof(int), hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_colptr);
    hipFree(d_indices);
    hipFree(d_states);
  }
  
  std::cout << " Done" << std::endl;
}

void saveSparseMatCSC(size_t* colptr, int* indices){
  
  std::cout << "Saving Sparse Matrix to: " << MAT_PATH ;
  ensureDirExists(MAT_PATH);
  
  std::ofstream colptrFile(MAT_PATH + "/colptr.txt");
  saveArrayToFile(colptrFile, colptr, (size_t) N+1);
  colptrFile.close();

  std::ofstream indicesFile(MAT_PATH + "/indices.txt");
  saveArrayToFile(indicesFile, indices, colptr[N]);
  indicesFile.close();

  std::cout << " Done" << std::endl;
}

void cscToDense(size_t* colptr, int* indices, int** dense) {
    // Initialize dense matrix with zeros
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            dense[i][j]=0;
        }
    }

    // Fill dense matrix with non-zero values from CSC format
    for (int j = 0; j < N; j++) {
        for (size_t i = colptr[j]; i < colptr[j+1]; i++) {
            dense[indices[i]][j] = 1 ;
        }
    }
}

int main(int argc , char** argv) {
  
  loadConfig("../conf/config_EI.yml");
  std::cout << " Done" << std::endl;
  
  size_t* colptr = new size_t[N+1]();
  int* indices = new int[(size_t) (N * N)]();

  for(int i_pop=0; i_pop<N_POP;++i_pop)
    for(int j_pop=0; j_pop<N_POP;++j_pop) {
      genSparseMatCSC(colptr, indices, Na[i_pop], Na[j_pop], Ka[j_pop], KAPPA[j_pop + N_POP * i_pop], cNa[j_pop]);
      
      std::cout << colptr[0] << " " << colptr[1] << std::endl;
    }
  
  if(IF_SAVE_MAT)
    saveSparseMatCSC(colptr, indices);
  
  // Remember to free dynamically allocated memory after you're done
  delete[] colptr;
  delete[] indices;
  
  return 0;
}
